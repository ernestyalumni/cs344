#include "hip/hip_runtime.h"
/* Udacity Homework 3
   HDR Tone-mapping

  Background HDR
  ==============

  A High Dynamic Range (HDR) image contains a wider variation of intensity
  and color than is allowed by the RGB format with 1 byte per channel that we
  have used in the previous assignment.  

  To store this extra information we use single precision floating point for
  each channel.  This allows for an extremely wide range of intensity values.

  In the image for this assignment, the inside of church with light coming in
  through stained glass windows, the raw input floating point values for the
  channels range from 0 to 275.  But the mean is .41 and 98% of the values are
  less than 3!  This means that certain areas (the windows) are extremely bright
  compared to everywhere else.  If we linearly map this [0-275] range into the
  [0-255] range that we have been using then most values will be mapped to zero!
  The only thing we will be able to see are the very brightest areas - the
  windows - everything else will appear pitch black.

  The problem is that although we have cameras capable of recording the wide
  range of intensity that exists in the real world our monitors are not capable
  of displaying them.  Our eyes are also quite capable of observing a much wider
  range of intensities than our image formats / monitors are capable of
  displaying.

  Tone-mapping is a process that transforms the intensities in the image so that
  the brightest values aren't nearly so far away from the mean.  That way when
  we transform the values into [0-255] we can actually see the entire image.
  There are many ways to perform this process and it is as much an art as a
  science - there is no single "right" answer.  In this homework we will
  implement one possible technique.

  Background Chrominance-Luminance
  ================================

  The RGB space that we have been using to represent images can be thought of as
  one possible set of axes spanning a three dimensional space of color.  We
  sometimes choose other axes to represent this space because they make certain
  operations more convenient.

  Another possible way of representing a color image is to separate the color
  information (chromaticity) from the brightness information.  There are
  multiple different methods for doing this - a common one during the analog
  television days was known as Chrominance-Luminance or YUV.

  We choose to represent the image in this way so that we can remap only the
  intensity channel and then recombine the new intensity values with the color
  information to form the final image.

  Old TV signals used to be transmitted in this way so that black & white
  televisions could display the luminance channel while color televisions would
  display all three of the channels.
  

  Tone-mapping
  ============

  In this assignment we are going to transform the luminance channel (actually
  the log of the luminance, but this is unimportant for the parts of the
  algorithm that you will be implementing) by compressing its range to [0, 1].
  To do this we need the cumulative distribution of the luminance values.

  Example
  -------

  input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
  min / max / range: 0 / 9 / 9

  histo with 3 bins: [4 7 3]

  cdf : [4 11 14]


  Your task is to calculate this cumulative distribution by following these
  steps.

*/

//#include "reference_calc.cpp"


#include <limits.h>
#include <float.h>
#include <math.h>
#include <stdio.h>

#include "utils.h"

__global__
void histogram_kernel(unsigned int* d_bins, const float* d_in, const int bin_count, const float lum_min, const float lum_max, const int size) {  
    int mid = threadIdx.x + blockDim.x * blockIdx.x;
    if(mid >= size)
        return;
    float lum_range = lum_max - lum_min;
    int bin = ((d_in[mid]-lum_min) / lum_range) * bin_count;
    
    atomicAdd(&d_bins[bin], 1);
}



__global__ 
void scan_kernel(unsigned int* d_bins, int size) {
    int mid = threadIdx.x + blockDim.x * blockIdx.x;
    if(mid >= size)
        return;
    
    for(int s = 1; s <= size; s *= 2) {
          int spot = mid - s; 
         
          unsigned int val = 0;
          if(spot >= 0)
              val = d_bins[spot];
          __syncthreads();
          if(spot >= 0)
              d_bins[mid] += val;
          __syncthreads();

    }
}

// Hillis Steele Scan - described in lecture
__global__ void cdf_kernel(unsigned int * d_in, const size_t numBins)
{
  int myId = threadIdx.x;
  for (int d = 1; d < numBins; d *= 2) {
    if ((myId + 1) % (d * 2) == 0) {
      d_in[myId] += d_in[myId - d];
    }
    __syncthreads();
  }
  if (myId == numBins - 1) d_in[myId] = 0;
  for (int d = numBins / 2; d >= 1; d /= 2) {
    if ((myId + 1) % (d * 2) == 0) {
      unsigned int tmp = d_in[myId - d];
      d_in[myId - d] = d_in[myId];
      d_in[myId] += tmp;
    }
    __syncthreads();
  }
}


// Blelloch Scan - described in lecture
__global__ void cdf_kernel_2(unsigned int * d_in, const size_t numBins)
{
  int myId = threadIdx.x;
  extern __shared__ float sdata[];
  sdata[myId] = d_in[myId];
  __syncthreads();            // make sure entire block is loaded!

  for (int d = 1; d < numBins; d *= 2) {
    if (myId >= d) {
      sdata[myId] += sdata[myId - d];
    }
    __syncthreads();
  }
  if (myId == 0)  d_in[0] = 0;
  else  d_in[myId] = sdata[myId - 1]; //inclusive->exclusive
}




// calculate reduce max or min and stick the value in d_answer.
__global__
void reduce_minmax_kernel(const float* const d_in, float* d_out, const size_t size, int minmax) {
    extern __shared__ float shared[];
    
    int mid = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x; 
    
    // we have 1 thread per block, so copying the entire block should work fine
    if(mid < size) {
        shared[tid] = d_in[mid];
    } else {
        if(minmax == 0)
            shared[tid] = FLT_MAX;
        else
            shared[tid] = -FLT_MAX;
    }
    
    // wait for all threads to copy the memory
    __syncthreads();
    
    // don't do any thing with memory if we happen to be far off ( I don't know how this works with
    // sync threads so I moved it after that point )
    if(mid >= size) {   
        if(tid == 0) {
            if(minmax == 0) 
                d_out[blockIdx.x] = FLT_MAX;
            else
                d_out[blockIdx.x] = -FLT_MAX;

        }
        return;
    }
       
    for(unsigned int s = blockDim.x/2; s > 0; s /= 2) {
        if(tid < s) {
            if(minmax == 0) {
                shared[tid] = min(shared[tid], shared[tid+s]);
            } else {
                shared[tid] = max(shared[tid], shared[tid+s]);
            }
        }
        
        __syncthreads();
    }
    
    if(tid == 0) {
        d_out[blockIdx.x] = shared[0];
    }
}


int get_max_size(int n, int d) {
    return (int)ceil( (float)n/(float)d ) + 1;
}

float reduce_minmax(const float* const d_in, const size_t size, int minmax) {
    int BLOCK_SIZE = 32;
    // we need to keep reducing until we get to the amount that we consider 
    // having the entire thing fit into one block size
    size_t curr_size = size;
    float* d_curr_in;
    
    checkCudaErrors(hipMalloc(&d_curr_in, sizeof(float) * size));    
    checkCudaErrors(hipMemcpy(d_curr_in, d_in, sizeof(float) * size, hipMemcpyDeviceToDevice));


    float* d_curr_out;
    
    dim3 thread_dim(BLOCK_SIZE);
    const int shared_mem_size = sizeof(float)*BLOCK_SIZE;
    
    while(1) {
        checkCudaErrors(hipMalloc(&d_curr_out, sizeof(float) * get_max_size(curr_size, BLOCK_SIZE)));
        
        dim3 block_dim(get_max_size(size, BLOCK_SIZE));
        reduce_minmax_kernel<<<block_dim, thread_dim, shared_mem_size>>>(
            d_curr_in,
            d_curr_out,
            curr_size,
            minmax
        );
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

            
        // move the current input to the output, and clear the last input if necessary
        checkCudaErrors(hipFree(d_curr_in));
        d_curr_in = d_curr_out;
        
        if(curr_size <  BLOCK_SIZE) 
            break;
        
        curr_size = get_max_size(curr_size, BLOCK_SIZE);
    }
    
    // theoretically we should be 
    float h_out;
    hipMemcpy(&h_out, d_curr_out, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_curr_out);
    return h_out;
}





__global__ void shmem_reduce_max_kernel( const float * d_in, float * d_out) {

	// sdata is allocated in the kernel call: 3rd arg to <<<b, t, shmem>>>
	extern __shared__ float sdata[];
	
	int k_x = threadIdx.x + blockDim.x * blockIdx.x ;
	int tid = threadIdx.x ;
	
	// load shared mem from global mem
	sdata[tid] = d_in[k_x] ;
	
//	if (k_x >= L) { return; }
	
	__syncthreads();			// make sure entire block is loaded!
	
	// do reduction in shared mem
	int M_x = blockDim.x; // M_x := total number of threads in a (single thread) block, ARRAY_SIZE = 32 in this case
	for (unsigned int s = M_x >> 1; s > 0; s >>= 1)
	{
		if (tid < s)
		{
			sdata[tid] = fmaxf( sdata[tid], sdata[tid + s] ) ;
		}
		__syncthreads() ;			// make sure all adds at one stage are done!
	}
	
	
	// only thread 0 writes result for this block back to global mem
	if (tid == 0)
	{
		d_out[blockIdx.x] = sdata[0];
	}
}


__global__ void shmem_reduce_min_kernel( const float * d_in, float * d_out) {

	// sdata is allocated in the kernel call: 3rd arg to <<<b, t, shmem>>>
	extern __shared__ float sdata[];
	
	int k_x = threadIdx.x + blockDim.x * blockIdx.x ;
	int tid = threadIdx.x ;
	
	// load shared mem from global mem
	sdata[tid] = d_in[k_x] ;
	
//	if (k_x >= L) { return; }

	__syncthreads();			// make sure entire block is loaded!
	
		
	// do reduction in shared mem
	int M_x = blockDim.x; // M_x := total number of threads in a (single thread) block, ARRAY_SIZE = 32 in this case
	for (unsigned int s = M_x >> 1; s > 0; s >>= 1)
	{
		if (tid < s)
		{
			sdata[tid] = fminf( sdata[tid], sdata[tid + s] ) ;
		}
		__syncthreads() ;			// make sure all adds at one stage are done!
	}
	
	
	// only thread 0 writes result for this block back to global mem
	if (tid == 0)
	{
		d_out[blockIdx.x] = sdata[0];
	}
}



void reduce_shmem_max(const float * d_in, float & out, const int L, int M_in) 
{
	int N_x = ( L + M_in - 1)/ M_in  ; 
	int M_x = M_in ;
	
	// declare GPU memory pointers
	float *dev_intermediate, *dev_out;
	
	// allocate GPU memory
	checkCudaErrors( 
		hipMalloc((void **) &dev_out, sizeof(float)) );
	checkCudaErrors( 
		hipMalloc((void **) &dev_intermediate, N_x * sizeof(float)) );
	
	shmem_reduce_max_kernel<<<N_x, M_x, M_x*sizeof(float)>>>( d_in, dev_intermediate ) ;
	
	// now we're down to one block left, so reduce it
	M_x = N_x;
	N_x = 1;

	shmem_reduce_max_kernel<<<N_x,M_x, M_x*sizeof(float)>>>( dev_intermediate, dev_out  ) ;

		// copy our results from device to host 
	checkCudaErrors(
		hipMemcpy( &out, dev_out, sizeof(float), hipMemcpyDeviceToHost) );

	checkCudaErrors( hipFree( dev_out ) );
	checkCudaErrors( hipFree( dev_intermediate ) );
}

//void reduce_shmem_min(float * d_in, float * out, const int L, int M_in) 
void reduce_shmem_min(const float * d_in, float & out, const int L, int M_in) 
{
	int N_x = ( L + M_in - 1)/ M_in  ; 
	int M_x = M_in ;
	
	// declare GPU memory pointers
	float *dev_intermediate, *dev_out;
	
	// allocate GPU memory
	checkCudaErrors( 
		hipMalloc((void **) &dev_out, sizeof(float)) );
	checkCudaErrors( 
		hipMalloc((void **) &dev_intermediate, N_x * sizeof(float)) );
	
	shmem_reduce_min_kernel<<<N_x, M_x, M_x*sizeof(float)>>>( d_in, dev_intermediate ) ;
	
	// now we're down to one block left, so reduce it
	M_x = N_x;
	N_x = 1;

	shmem_reduce_min_kernel<<<N_x,M_x, M_x*sizeof(float)>>>( dev_intermediate, dev_out  ) ;

		// copy our results from device to host 
	checkCudaErrors(
//		hipMemcpy( out, dev_out, sizeof(float), hipMemcpyDeviceToHost) );
		hipMemcpy( &out, dev_out, sizeof(float), hipMemcpyDeviceToHost) );


	checkCudaErrors( hipFree( dev_out ) );
	checkCudaErrors( hipFree( dev_intermediate ) );
}


__global__ void histo_kernel(unsigned int * d_out, const float * const d_in,
  const size_t numBins, float logLumRange, float min_logLum)
{
  int myId = threadIdx.x + blockDim.x * blockIdx.x;
  int bin = (d_in[myId] - min_logLum) / logLumRange * numBins;
  if (bin == numBins)  bin--;
  atomicAdd(&d_out[bin], 1);
}

__device__ unsigned int normalize_lum(const float x_in, float max_logLum, float min_logLum, const unsigned int numBins) {
	float logLumRange = max_logLum - min_logLum; 
	unsigned int bin = ( x_in - min_logLum) / logLumRange * numBins ; 
	return bin ; 
}

__global__ void histogram_shared( const float* const d_vals, unsigned int *d_Histo, 
	const unsigned int numBins, const unsigned int numElems,
	float & max_logLum, float & min_logLum ) {
		
	int k_x = threadIdx.x + blockDim.x * blockIdx.x ; // k_x = 0, 1, ... numElems-1 
	int i_x = threadIdx.x ; 
	int M_x = blockDim.x  ; 
	int offset = blockDim.x * gridDim.x ; 
	
	extern __shared__ unsigned int s[] ; // |s| = numBins ; i.e. size of s, shared memory, is numBins

/*	
	for (unsigned int i = i_x; i < numBins; i += M_x ) {
		s[i] = 0; }
*/
	if (i_x < numBins) { s[i_x] = 0; }
	__syncthreads(); 	

	if (k_x >= numElems) { return ; }
	
/*	for (unsigned int i = k_x; i < numElems; i += offset) {
		unsigned int bin = normalize_lum( d_vals[i] , max_logLum, min_logLum, numBins );

		atomicAdd( &s[ bin ], 1) ;
	}*/

	
	unsigned int bin = normalize_lum( d_vals[k_x] , max_logLum, min_logLum, numBins );
	atomicAdd( &s[ bin ], 1) ;

	
	__syncthreads(); // ensure last of our writes have been committed

/*
	for (unsigned int i = i_x; i < numBins; i += M_x ) {
		atomicAdd( &(d_Histo[ i ]), s[ i ] ) ;
	}
*/
	atomicAdd( &(d_Histo[ i_x ]), s[ i_x ] ) ;

	__syncthreads(); 	

		
}



void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
  //TODO
  /*Here are the steps you need to implement
    1) find the minimum and maximum value in the input logLuminance channel
       store in min_logLum and max_logLum
    2) subtract them to find the range
    3) generate a histogram of all the values in the logLuminance channel using
       the formula: bin = (lum[i] - lumMin) / lumRange * numBins
    4) Perform an exclusive scan (prefix sum) on the histogram to get
       the cumulative distribution of luminance values (this should go in the
       incoming d_cdf pointer which already has been allocated for you)       */


 const size_t size = numRows*numCols;
//    min_logLum = reduce_minmax(d_logLuminance, size, 0);
 //   max_logLum = reduce_minmax(d_logLuminance, size, 1);

    reduce_shmem_max(d_logLuminance, max_logLum, size, 1024) ; 
    reduce_shmem_min(d_logLuminance, min_logLum, size, 1024) ; 
    
    printf("got min of %f\n", min_logLum);
    printf("got max of %f\n", max_logLum);
    printf("numBins %d\n", numBins);
    
//    unsigned int* d_bins;
    size_t histo_size = sizeof(unsigned int)*numBins;

//    checkCudaErrors(hipMalloc(&d_bins, histo_size));    
 //   checkCudaErrors(hipMemset(d_bins, 0, histo_size));  
    dim3 thread_dim(1024);
    dim3 hist_block_dim(get_max_size(size, thread_dim.x));
    
    histogram_kernel<<<hist_block_dim, thread_dim>>>(d_cdf, d_logLuminance, numBins, min_logLum, max_logLum, size);
    
    //histogram_kernel<<<hist_block_dim, thread_dim>>>(d_bins, d_logLuminance, numBins, min_logLum, max_logLum, size);
    
    
/*    
    float logLumRange = max_logLum - min_logLum ; 
histo_kernel<<<(size + thread_dim.x - 1)/thread_dim.x, thread_dim.x >>>( d_cdf, d_logLuminance,
  numBins, logLumRange, min_logLum) ;
    
  */  
  
 /*   
	histogram_shared<<<(size + thread_dim.x - 1)/thread_dim.x, thread_dim.x, numBins*sizeof(unsigned int)>>>( 
		d_logLuminance, d_cdf, numBins, size,  max_logLum, min_logLum ) ;
   */ 
    
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

/*    unsigned int h_out[100];
    hipMemcpy(&h_out, d_bins, sizeof(unsigned int)*100, hipMemcpyDeviceToHost);
    for(int i = 0; i < 100; i++)
//        printf("hist out %d\n", h_out[i]);
          printf("hist out %d ", h_out[i]);
  */
    
    dim3 scan_block_dim(get_max_size(numBins, thread_dim.x));

//    scan_kernel<<<scan_block_dim, thread_dim>>>(d_bins, numBins);

// cdf_kernel_2 << <(numBins + numBins - 1)/numBins, numBins, sizeof(unsigned int)* numBins >> >(d_bins, numBins);
 cdf_kernel_2 << <(numBins + numBins - 1)/numBins, numBins, sizeof(unsigned int)* numBins >> >(d_cdf, numBins);


    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

/*    
    hipMemcpy(&h_out, d_bins, sizeof(unsigned int)*100, hipMemcpyDeviceToHost);
    for(int i = 0; i < 100; i++)
//        printf("cdf out %d\n", h_out[i]);
          printf("cdf out %d ", h_out[i]);
  */  

//    hipMemcpy(d_cdf, d_bins, histo_size, hipMemcpyDeviceToDevice);

    
//checkCudaErrors(hipFree(d_bins));

}
