#include "hip/hip_runtime.h"
/* main.cu
 * Ernest Yeung
 * ernestyalumni@gmail.com
 * Demonstrates Hillis/Steele and Blelloch (exclusive) scan with a parallel implementation
 * with CUDA C/C++ and global memory
 * 
 * */
#include <vector> // std::vector
#include <algorithm> // std::fill
#include <cmath> // std::log2
#include <chrono> // chrono::steady_clock::now() 

#include "./common/timer.h"  // GpuTimer

#include "./methods/checkerror.h"
#include "./methods/scans.h" /* Blelloch_up_global, Blelloch_down_global, copy_swap
								* Blelloch_scan_kernelLauncher, 
								* HillisSteele_global, HillisSteele_kernelLauncher */

int main() {
	// "boilerplate"
	// initiate correct GPU
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	if (deviceCount == 0) {
		exit(EXIT_FAILURE);
	}
	int dev = 0;
	hipSetDevice(dev);
	
	hipDeviceProp_t devProps;
	if (hipGetDeviceProperties(&devProps, dev) == 0) {
		std::cout << " Using device " << dev << ":\n" ;
		std::cout << devProps.name << "; global mem: " << (int)devProps.totalGlobalMem <<
			"; compute v" << (int)devProps.major << "." << (int)devProps.minor << "; clock: " <<
			(int)devProps.clockRate << " kHz" << std::endl; }
	// END if GPU properties
	
	// MANUALLY CHANGE THESE 2 : ARRAY_SIZE, DISPLAY_SIZE
	// input array with interesting values "boilerplate"
	const int ARRAY_SIZE { 1<< 18};
//	const int ARRAY_SIZE { 1<<20 } ;  
									/* 2^20=1048576 worked on GTX 980 Ti; for larger, such as 2^22, Segmentation Fault
										* for only the Blelloch scan.  
										* For Hillis/Steele, 2^20=1048576 worked on GTX 980 Ti; for larger, such as 2^21, 
										* CUDA error
										* Note that it was critical that M_x is maxed out (on GTX 980 Ti, 
										* it's 1024 max. number of threads per block
										* */
//	const long ARRAY_SIZE { 1<< 29 }; // this this line if needed
	const int ARRAY_BYTES { ARRAY_SIZE * sizeof(float) } ;
//	const long ARRAY_BYTES { ARRAY_SIZE * sizeof(float) }; // use this line if needed
	const int L_x { ARRAY_SIZE } ;
	std::cout << "For an (float) array of size (length) : " << ARRAY_SIZE << std::endl ;
	std::cout << "or, in bytes, " << ARRAY_BYTES << std::endl;
	
	
	const int DISPLAY_SIZE = 22; // how many numbers you want to display, read out, or print out on screen
	static_assert( ARRAY_SIZE >= DISPLAY_SIZE, "ARRAY_SIZE needs to be equal or bigger than DISPLAY_SIZE");
	
	// generate input array on host
	std::vector<float> f_vec;
	for (int i = 0; i < ARRAY_SIZE; ++i) {
		f_vec.push_back(i+1) ; }
	float* host_f_in;
	host_f_in = f_vec.data();

/*	std::vector<unsigned int> in_vec;
	for (int i = 0; i < ARRAY_SIZE; ++i) {
		in_vec.push_back(i+1); }
*/
/*
	std::vector<unsigned int> in_vec(ARRAY_SIZE);
	std::fill(in_vec.begin(),in_vec.end(),1); 
*/
	std::vector<unsigned int> in_vec;
	for (int i = 0; i < ARRAY_SIZE; ++i) {
		in_vec.push_back(1); }

	// sanity check print out of initial values: 
	std::cout << " Initially, " << std::endl;
	for (int i = 0 ; i < DISPLAY_SIZE; ++i)  {
		std::cout << " " << f_vec[i] ; }
	std::cout << std::endl;

	std::cout << " in_vec : " << std::endl;
	for (int i = 0 ; i < DISPLAY_SIZE; ++i)  {
		std::cout << " " << in_vec[i] ; }
	std::cout << std::endl;


	// END of initializing, creating input array with interesting values, on host CPU, boilerplate
	///////////////////////////////////////////////////////////////////

	// declare GPU memory pointers
	float *dev_f_in, *dev_f_out;
	unsigned int* d_in; 
	//, d_in2;
//	unsigned int* d_out;
	
	// allocate GPU memory
	checkCudaErrors(
		hipMalloc((void **) &dev_f_in, ARRAY_BYTES ));
	checkCudaErrors(
		hipMalloc((void **) &dev_f_out, ARRAY_BYTES));

	checkCudaErrors(
		hipMalloc((void **) &d_in, ARRAY_SIZE*sizeof(unsigned int)));
//	checkCudaErrors(
	//	hipMalloc((void **) &d_in2, ARRAY_SIZE*sizeof(unsigned int)));

//	checkCudaErrors(
//		hipMalloc((void **) &d_out, ARRAY_SIZE*sizeof(unsigned int)));


	// transfer the input array to the GPU
	checkCudaErrors(
		hipMemcpy(dev_f_in, host_f_in, ARRAY_BYTES, hipMemcpyHostToDevice) );
	checkCudaErrors(
		hipMemcpy(d_in, in_vec.data(), ARRAY_SIZE*sizeof(unsigned int), hipMemcpyHostToDevice) );
//	checkCudaErrors(
//		hipMemcpy(d_in2, in2_vec.data(), ARRAY_SIZE*sizeof(unsigned int), hipMemcpyHostToDevice));
	
	
	///////////////////////////////////////////////////////////////////
	// grid, block dimensions
	///////////////////////////////////////////////////////////////////
	// MANUALLY CHANGE THIS 1: M_x
	// launch the kernel
	// input parameters for number of threads per block, M_x, and number of blocks, N_x
	const int M_x {1024};
	
	// END of grid, block dimensions
	///////////////////////////////////////////////////////////////////
	
	
	// Blelloch scan (exclusive scan) in parallel
	// time the kernel
	GpuTimer timer;
	timer.Start();

	Blelloch_scan_kernelLauncher(dev_f_in, dev_f_out, L_x, M_x) ;

	timer.Stop();
	
	hipDeviceSynchronize(); 
	checkCudaErrors(
		hipGetLastError() );
		
	std::cout<< "Blelloch scan, in parallel, ran in : " << timer.Elapsed() << " msecs. " << std::endl;
	
	std::cout << "After Blelloch scan : " << std::endl;
		
	// copy our results from device to host
	float host_f_out[ARRAY_SIZE];
	checkCudaErrors( hipMemcpy(host_f_out, dev_f_in, ARRAY_BYTES, hipMemcpyDeviceToHost) );
	// read out results into our useful vector
	f_vec.insert(f_vec.begin(), &host_f_out[0], &host_f_out[ARRAY_SIZE] );
	// print out results
	for (int i = 0 ; i < DISPLAY_SIZE; ++i)  {
		std::cout << " " << f_vec[i] ; }
	std::cout << std::endl;
	// uncomment for printout of previous step
/*	checkCudaErrors( 
		hipMemcpy(host_f_out, dev_f_out, ARRAY_BYTES, hipMemcpyDeviceToHost) );
	f_vec.insert(f_vec.begin(), &host_f_out[0], &host_f_out[ARRAY_SIZE] );
	for (int i = 0 ; i < DISPLAY_SIZE; ++i)  {
		std::cout << " " << f_vec[i] ; }  */
	std::cout << std::endl;

	/** ****************************************************************
	 * ** Blelloch scan (exclusive scan) in parallel for any field T ***
	 * ****************************************************************/
	// time the Blelloch scan, for unsigned int
	timer.Start();

//	Blelloch_scan_kernelLauncher<unsigned int>(d_in, d_out, L_x, M_x) ;
	Blelloch_scan_kernelLauncher<unsigned int,0>(d_in, L_x, M_x) ;


	timer.Stop();

	hipDeviceSynchronize(); 
	checkCudaErrors(
		hipGetLastError() );

	std::cout<< "\n Blelloch scan for unsigned int, in parallel, ran in : " << 
		timer.Elapsed() << " msecs. " << std::endl;
	
	std::cout << "\n After Blelloch scan for unsigned int : " << std::endl;

		// copy our results from device to host
	checkCudaErrors( 
		hipMemcpy(in_vec.data(), d_in, 
			ARRAY_SIZE*sizeof(unsigned int), hipMemcpyDeviceToHost) );

	// print out results
	for (int i = 0 ; i < DISPLAY_SIZE; ++i)  {
		std::cout << " " << in_vec[i] ; }
	std::cout << std::endl;

	for (auto iter = in_vec.end()-DISPLAY_SIZE ; iter < in_vec.end(); ++iter)  {
		std::cout << " " << *iter ; }
	std::cout << std::endl;


/*
	Blelloch_scan_kernelLauncher<unsigned int,0>(d_in2, L_x, M_x) ;
	std::cout << "\n After Blelloch scan for unsigned int, all 1's : " << std::endl;

		// copy our results from device to host
	checkCudaErrors( 
		hipMemcpy(in2_vec.data(), d_in2, 
			ARRAY_SIZE*sizeof(unsigned int), hipMemcpyDeviceToHost) );

	// print out results
	for (int i = 0 ; i < DISPLAY_SIZE; ++i)  {
		std::cout << " " << in2_vec[i] ; }
	std::cout << std::endl;

	for (auto iter = in2_vec.end()-DISPLAY_SIZE ; iter < in2_vec.end(); ++iter)  {
		std::cout << " " << *iter ; }
	std::cout << std::endl;
*/


	/** END of Blelloch scan (exclusive scan) in parallel for any field T */
	////////////////////////////////////////////////////////////////////

	// Hillis/Steele scan, in parallel
	// transfer the input array to the GPU
	for (int i = 0; i < ARRAY_SIZE; ++i) {
		f_vec[i] = i  ; }
	host_f_in = f_vec.data();

	hipMemcpy(dev_f_in, host_f_in, ARRAY_BYTES, hipMemcpyHostToDevice);

	// time the Hillis/Steele scan, in parallel, kernel
	timer.Start();

	HillisSteele_kernelLauncher(dev_f_in, dev_f_out, L_x, M_x) ;

	timer.Stop();
	
	hipDeviceSynchronize(); 
	checkCudaErrors(
		hipGetLastError() );
	
	std::cout<< "Hillis/Steele scan, in parallel, ran in : " << timer.Elapsed() << " msecs. " << std::endl;
	
	std::cout << "After Hillis/Steele scan : " << std::endl;

		// copy our results from device to host
	checkCudaErrors( hipMemcpy(host_f_out, dev_f_in, ARRAY_BYTES, hipMemcpyDeviceToHost) );
	// read out results into our useful vector
	f_vec.insert(f_vec.begin(), &host_f_out[0], &host_f_out[ARRAY_SIZE] );
	// print out results
	for (int i = 0 ; i < DISPLAY_SIZE; ++i)  {
		std::cout << " " << f_vec[i] ; }
	std::cout << std::endl;
	// uncomment for printout of previous step
/*	checkCudaErrors( 
		hipMemcpy(host_f_out, dev_f_out, ARRAY_BYTES, hipMemcpyDeviceToHost) );
	f_vec.insert(f_vec.begin(), &host_f_out[0], &host_f_out[ARRAY_SIZE] );
	for (int i = 0 ; i < DISPLAY_SIZE; ++i)  {
		std::cout << " " << f_vec[i] ; } */
	std::cout << std::endl;

	// Blelloch scan, in serial
	// vector for serial implementation
//	std::vector<float> f_vec_out( ARRAY_SIZE,0); // I obtain error : name followed by "::" must be a class or namespace name
//          detected during:
 //           instantiation of class "std::__iterator_traits<_Iterator, void> [with _Iterator=int]" 

//	std::vector<float> f_vec_out;

	for (int i = 0; i < ARRAY_SIZE; ++i) {
		f_vec[i] = ((float) (i+1))  ; }	
	// sanity check print out 
	std::cout << " For the serial implementation of Blelloch scan, initially : " << std::endl;
	for (int i = 0; i <DISPLAY_SIZE;++i) {
		std::cout << f_vec[i] << " " ; }
	std::cout << std::endl;

	// For measuring execution time of a piece of code, use now() function of chrono's steady_clock
	auto start = std::chrono::steady_clock::now(); 

	blelloch_serial( f_vec);

	auto end = std::chrono::steady_clock::now(); 

	// print out results
	for (int i = 0 ; i < DISPLAY_SIZE; ++i)  {
		std::cout << f_vec[i] << " " ; } 
	std::cout << std::endl;

/*	for (int i = 0 ; i < DISPLAY_SIZE; ++i)  {
		std::cout << " " << f_vec_out[i] ; } 
	std::cout << std::endl;
*/
	for (auto iter = f_vec.end() - DISPLAY_SIZE ; iter < f_vec.end(); ++iter)  {
		std::cout << *iter << " " ; } 
	std::cout << std::endl;


	auto diff = end - start;
	std::cout << " Blelloch scan, in serial, ran in : " << 
		std::chrono::duration <double, std::milli>(diff).count() << " ms " << std::endl ; 

	// Hillis-Steele (inclusion) scan, serial
	for (int i = 0; i < ARRAY_SIZE; ++i) {
		f_vec[i] = ((float) i )  ; }	
	// sanity check print out 
	std::cout << " For the serial implementation of Hillis-Steele scan, initially : " << std::endl;
	for (int i = 0; i <DISPLAY_SIZE;++i) {
		std::cout << f_vec[i] << " " ; }
	std::cout << std::endl;


	start = std::chrono::steady_clock::now(); 
	HillisSteele_serial( f_vec );
	end = std::chrono::steady_clock::now(); 

	// print out results
	for (int i = 0 ; i < DISPLAY_SIZE; ++i)  {
		std::cout << f_vec[i] << " " ; } 
	std::cout << std::endl;
	
	diff = end - start;
	std::cout << " Hillis-Steele scan, in serial, ran in : " << 
		std::chrono::duration <double, std::milli>(diff).count() << " ms " << std::endl ; 
	


	// free GPU memory
	checkCudaErrors( hipFree( dev_f_in  ) );
	checkCudaErrors( hipFree( dev_f_out ) );

	checkCudaErrors( hipFree( d_in ) );
//	checkCudaErrors( hipFree( d_out ) );


}
