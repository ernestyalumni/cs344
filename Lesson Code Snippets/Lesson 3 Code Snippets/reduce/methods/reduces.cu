#include "hip/hip_runtime.h"
/* reduces.cu
 * Ernest Yeung
 * ernestyalumni@gmail.com
 * Demonstrates reduce with parallel and serial implementations
 * with CUDA C/C++ and global memory
 * 
 * */
#include "reduces.h"

// parallel implementations

__global__ void global_reduce_kernel( float * d_in, float * d_out, const int L ) 
{
	int k_x = threadIdx.x + blockDim.x * blockIdx.x ; 
	int tid = threadIdx.x ;
	
	if (k_x >= L) {
		return; }
	
	// do reduction in global mem
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (tid < s)
		{
			d_in[k_x] += d_in[k_x + s];
		}
		__syncthreads(); 		// make sure all adds at one stage are done!
	}
	
	// only thread 0 writes result for this block back to global mem
	if (tid == 0) 
	{
		d_out[blockIdx.x] = d_in[k_x];
	}
}

__global__ void shmem_reduce_kernel(const float * d_in, float * d_out, const int L )
{
	// sdata is allocated in the kernel call: 3rd arg to <<<b, t, shmem>>>
	extern __shared__ float sdata[];
	
	int k_x = threadIdx.x + blockDim.x * blockIdx.x ;
	int tid = threadIdx.x ;
	
	// load shared mem from global mem
	sdata[tid] = d_in[k_x] ;
	__syncthreads();			// make sure entire block is loaded!
	
	if (k_x >= L ) {
		return; }
	
	
	// do reduction in shared mem
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (tid < s)
		{
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads() ;			// make sure all adds at one stage are done!
	}
	
	
	// only thread 0 writes result for this block back to global mem
	if (tid == 0)
	{
		d_out[blockIdx.x] = sdata[0];
	}
}

__global__ void shmem_reduce_add_kernel(const float* d_in, float* d_out, const int L ) {
	// sdata is allocated in the kernel call: 3rd arg to <<<b, t, shmem>>>
	extern __shared__ float sdata[];
	
	int k_x = threadIdx.x + blockDim.x + blockIdx.x ;
	
	
}

void reduce_global(float * d_in, float * out, const int L, int M_in) 
{
	int N_x { ( L + M_in - 1)/ M_in } ; 
	int M_x { M_in };
	
	
	// declare GPU memory pointers
	float *dev_intermediate, *dev_out;
	
	// allocate GPU memory
	checkCudaErrors( 
		hipMalloc((void **) &dev_out, sizeof(float)) );
	checkCudaErrors( 
		hipMalloc((void **) &dev_intermediate, N_x * sizeof(float)) );
	
	global_reduce_kernel<<<N_x, M_x>>>( d_in, dev_intermediate, L ) ;
	
	// now we're down to one block left, so reduce it
	M_x = N_x;
	N_x = 1;

	global_reduce_kernel<<<N_x,M_x>>>( dev_intermediate, dev_out, M_x) ;

		// copy our results from device to host 
	checkCudaErrors(
		hipMemcpy( out, dev_out, sizeof(float), hipMemcpyDeviceToHost) );

	hipFree( dev_out );
	hipFree( dev_intermediate );
}

void reduce_shmem(float * d_in, float * out, const int L, int M_in) 
{
	int N_x { ( L + M_in - 1)/ M_in } ; 
	int M_x { M_in };
	
	// declare GPU memory pointers
	float *dev_intermediate, *dev_out;
	
	// allocate GPU memory
	checkCudaErrors( 
		hipMalloc((void **) &dev_out, sizeof(float)) );
	checkCudaErrors( 
		hipMalloc((void **) &dev_intermediate, N_x * sizeof(float)) );
	
	shmem_reduce_kernel<<<N_x, M_x, M_x*sizeof(float)>>>( d_in, dev_intermediate, L ) ;
	
	// now we're down to one block left, so reduce it
	M_x = N_x;
	N_x = 1;

	shmem_reduce_kernel<<<N_x,M_x, M_x*sizeof(float)>>>( dev_intermediate, dev_out, M_x ) ;

		// copy our results from device to host 
	checkCudaErrors(
		hipMemcpy( out, dev_out, sizeof(float), hipMemcpyDeviceToHost) );

	hipFree( dev_out );
	hipFree( dev_intermediate );
}
